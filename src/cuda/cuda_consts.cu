#include "cuda_hash_params.h"
#include "hip/hip_runtime_api.h"
#include "cuda_raycast_params.h"

__constant__ HashParams c_hashParams;
__constant__ RayCastParams c_rayCastParams;
// __constant__ DepthCameraParams c_depthCameraParams;

void updateConstantHashParams(const HashParams& params) {

	size_t size;
	checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(c_hashParams)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_hashParams), &params, size, 0, hipMemcpyHostToDevice));
	
}

void updateConstantRayCastParams(const RayCastParams& params) {
	
	size_t size;
	checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(c_rayCastParams)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_rayCastParams), &params, size, 0, hipMemcpyHostToDevice));
	
}