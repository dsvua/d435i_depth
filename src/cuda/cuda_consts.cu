#include "cuda_hash_params.h"
#include "hip/hip_runtime_api.h"

__constant__ HashParams c_hashParams;
// __constant__ RayCastParams c_rayCastParams;
// __constant__ DepthCameraParams c_depthCameraParams;

void updateConstantHashParams(const HashParams& params) {

	size_t size;
	checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(c_hashParams)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_hashParams), &params, size, 0, hipMemcpyHostToDevice));
	
}

